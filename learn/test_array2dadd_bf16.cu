#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <unistd.h>
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>



__global__ void addMatrixKernel(__hip_bfloat16* A, __hip_bfloat16* B, __hip_bfloat16* C, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < numRows && col < numCols) {
        int index = row * numCols + col;
        C[index] = A[index] + B[index];
        printf("Thread (%d, %d) - A[%d, %d] + B[%d, %d] = %.1f + %.1f = %.1f\n", row, col, row, col, row, col, (float)A[index], (float)B[index], (float)C[index]);
        printf("blockIdx (%d, %d), blockDim (%d, %d), threadIdx (%d, %d)\n", blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    }
}


int main() {
    // Define matrix dimensions (2x2 matrices)
    int numRows = 4;  
    int numCols = 4;  
    int numElements = numRows * numCols;
    size_t size = numElements * sizeof(__hip_bfloat16);

    // Define and initialize host matrices
    __hip_bfloat16 h_A[16] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    __hip_bfloat16 h_B[16] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    __hip_bfloat16 h_C[16];  // Result matrix C

    // Allocate device arrays
    __hip_bfloat16 *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set execution configuration
    dim3 threadsPerBlock(2, 2);  // Block dimensions: 2x2, perfectly fits our matrix
    dim3 numBlocks(2, 2);        // Grid dimensions: 1x1, only one block needed

    // Launch kernel
    addMatrixKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, numRows, numCols);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Output the result
    std::cout << "Matrix C:" << std::endl;
    for (int i = 0; i < numElements; ++i) {
        std::cout << (float)h_C[i] << " ";
        if ((i + 1) % numCols == 0)
            std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
